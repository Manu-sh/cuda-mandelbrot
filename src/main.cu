#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <ppm3.hpp>
#include <Matrix1D.hpp>
#include <common.hpp>

#include <hip/hip_runtime.h>
#include <thrust/complex.h>

inline constexpr uint16_t cols = 1920, rows = 1080;
//inline constexpr uint16_t cols = 4096, rows = 3112;
static_assert(
        ((rows-1) * cols) <= 0xff'ff'ff,
        "ATfast(cols, r_idx, c_idx) -> __umul24(r_idx, cols) doesnt support more than 24 bit"
);


// https://stackoverflow.com/questions/16119923/using-constants-with-cuda
__device__ inline constexpr uint16_t ixsize = rows, gpu_rows = rows, iysize = cols, gpu_cols = cols, max_i = 1000;
__device__ inline constexpr float cxmin = -2.5f, cxmax = 2.5f, cymin = -2.5f, cymax = 2.5f;

#if 0
#include <hip/hip_fp16.h>

__device__ inline thrust::complex<float> fast_mul(thrust::complex<float> a, thrust::complex<float> b) {

    // c.real() = a.real() * b.real() - a.imag() * b.imag()
    // c.imag() = a.real() * b.imag() + a.imag() * b.real()

    using fp16 = __half;
    fp16 a_real = (fp16)a.real(), a_imag = (fp16)a.imag();
    fp16 b_real = (fp16)b.real(), b_imag = (fp16)b.imag();

    //fp16 c_real = a_real * b_real - a_imag * b_imag;
    //fp16 c_imag = a_real * b_imag + a_imag * b_real;

    fp16 c_real = __hmul(a_real, b_real) - __hmul(a_imag, b_imag);
    fp16 c_imag = __hmul(a_real, b_imag) + __hmul(a_imag, b_real);

    return {c_real, c_imag};
}

__device__ inline thrust::complex<float> ffast_mul(thrust::complex<float> a, thrust::complex<float> b) {

    // c.real() = a.real() * b.real() - a.imag() * b.imag()
    // c.imag() = a.real() * b.imag() + a.imag() * b.real()

    using fp16 = __half;
    using fp16_vector = __half2;

    fp16_vector a_vec = __halves2half2((fp16)a.real(), (fp16)a.imag()); // a[] = {a.real(), a.imag() }
    fp16_vector b_vec = __halves2half2((fp16)b.real(), (fp16)b.imag()); // b[] = {b.real(), b.imag() }

    // c[] = {a[0] * b[0], a[1] * b[1]}; -> a.real() * b_real(), a.imag() * b.imag()
    // c[] = {a.real() * b_real(), a.imag() * b.imag()};
    fp16_vector c_real_tmp = __hmul2(a_vec, b_vec); // computa le prime 2 moltiplicazioni necessarie a calcolare la parte real di c senza fare sottrazione

    // d[] = { a[0] * b[1], a[1] * b[0] }
     // fp16_vector c_imag_tmp = __hmul2(a_vec, __halves2half2(__high2half(b_vec), __low2half(b_vec)));
    fp16_vector c_imag_tmp = __halves2half2(
             __low2half(a_vec) * __high2half(b_vec), // d[0] = a[0] * b[1] -> a.real() * b.imag()
            __high2half(a_vec) *  __low2half(b_vec)  // d[1] = a[1] * b[0] -> a.imag() * b.real()
    );

    fp16_vector c_vec = __halves2half2(
            __low2half(c_real_tmp) - __high2half(c_real_tmp), // real part
            __low2half(c_imag_tmp) + __high2half(c_imag_tmp)  // imag part
    );

    return {__low2float(c_vec), __high2float(c_vec)};
}
#endif

__device__ rgb_t calc_mandelbrot(uint16_t ix, uint16_t iy) {

    using thrust::complex, thrust::abs, thrust::cos;

    complex<float> c{
            cxmin + ix / (ixsize-1.0f) * (cxmax - cxmin),
            cymin + iy / (iysize-1.0f) * (cymax - cymin)
    }, z = 0;

    c *= c * c; // c = pow(c, 3);

    uint16_t i;
    for (i = 0; i < max_i && abs(z) < 4.0f; ++i)
        z = cos(z * c); // z = cos(z * pow(c, 3));

    // GPU handle float truncation differently from CPU, making round-toward-zero instead of round-to-nearest.
    // that took me a lot debugging to figure why the image generated was so different than another generated
    // by the same code but running on cpu instead of gpu

    //const uint8_t col = (uint8_t)(int)(z.real() * 1);                 // -> works but is signed integer overflow which is UB
    //const uint8_t col = min(255, max(0, (unsigned)(z.real() * 255))); // -> different result
    //const uint8_t col = (uint8_t)lround(z.real() * 1);                // ok
    const uint8_t col = (uint8_t)__float2int_rn(z.real() * 1);          // same of calling std::lround(float) but cuda specific
    return (i == max_i) ? rgb_t{0,0,0} : rgb_t{col,col,col};
}


__global__ void kernel(rgb_t *const v, uint32_t len) {

          uint16_t tr = blockIdx.y * blockDim.y + threadIdx.y;
    const uint16_t tc = blockIdx.x * blockDim.x + threadIdx.x;

    for (; tr < gpu_rows; tr += blockDim.y * gridDim.y) {
        for (uint16_t c = tc; c < gpu_cols; c += blockDim.x * gridDim.x) {
            const uint32_t index = ATfast(gpu_cols, tr, c);
            //assert(index < len);
            v[index] = calc_mandelbrot(tr, c);
            //v[ __umul24(tr, gpu_cols) + c ] = calc_mandelbrot(tr, c);
        }
    }

}

int main() {

    hipSetDevice(0);

    rgb_t *gpu_vct;
    hipMalloc(&gpu_vct, sizeof(rgb_t) * cols * rows);

    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipMemset(gpu_vct, 0, sizeof(rgb_t) * cols * rows);

    dim3 threadsPerBlock((int)sqrt(maxThreadsPerBlock), (int)sqrt(maxThreadsPerBlock));
    dim3 blocksDim(
            (cols + threadsPerBlock.x - 1) / threadsPerBlock.x, // forza l'allineamento a un multiplo di threadsPerBlock
            (rows + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    kernel<<<blocksDim, threadsPerBlock>>>(gpu_vct, cols * rows);
    hipDeviceSynchronize(); // wait for gpu

    PPM3 img{cols, rows};
    hipMemcpy(img.unwrap(), gpu_vct, sizeof(PPM3::pixel_type) * img.width() * img.height(), hipMemcpyDeviceToHost);
    img.write_file_content("test.ppm");

    hipFree(gpu_vct);
    hipDeviceReset();
    return 0;
}
