#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#include <PPM.hpp>
#include <common.hpp>
#include <memalign/utils.hpp>

inline constexpr uint16_t cols = 1920, rows = 1080;
//inline constexpr uint16_t cols = 4096, rows = 3112;
static_assert(
        ((rows-1) * cols) <= 0xff'ff'ff,
        "ATfast(cols, r_idx, c_idx) -> __umul24(r_idx, cols) does not support more than 24 bit"
);


// https://stackoverflow.com/questions/16119923/using-constants-with-cuda
__device__ inline constexpr uint16_t ixsize = rows, gpu_rows = rows, iysize = cols, gpu_cols = cols, max_i = 1000;
__device__ inline constexpr float cxmin = -2.5f, cxmax = 2.5f, cymin = -2.5f, cymax = 2.5f;

__device__ rgb_t calc_mandelbrot(uint16_t ix, uint16_t iy) {

    using thrust::complex, thrust::abs, thrust::cos;

    complex<float> c{
            cxmin + ix / (ixsize-1.0f) * (cxmax - cxmin),
            cymin + iy / (iysize-1.0f) * (cymax - cymin)
    }, z = 0;

    c *= c * c; // c = pow(c, 3);

    uint16_t i;
    for (i = 0; i < max_i && abs(z) < 4.0f; ++i)
        z = cos(z * c); // z = cos(z * pow(c, 3));

    // GPU handle float truncation differently from CPU, making round-toward-zero instead of round-to-nearest.
    // that took me a lot debugging to figure why the image generated was so different than another generated
    // by the same code but running on cpu instead of gpu

    //const uint8_t col = (uint8_t)(int)(z.real() * 1);                 // -> works but is signed integer overflow which is UB
    //const uint8_t col = min(255, max(0, (unsigned)(z.real() * 255))); // -> different result
    //const uint8_t col = (uint8_t)lround(z.real() * 1);                // ok
    const uint8_t col = (uint8_t)__float2int_rn(z.real() * 1);          // same of calling std::lround(float) but cuda specific
    return (i == max_i) ? rgb_t{0,0,0} : rgb_t{col,col,col};
}


__global__ void kernel(rgb_t *const v, uint32_t len) {

          uint16_t tr = blockIdx.y * blockDim.y + threadIdx.y;
    const uint16_t tc = blockIdx.x * blockDim.x + threadIdx.x;

    #pragma unroll
    for (; tr < gpu_rows; tr += blockDim.y * gridDim.y) {
        #pragma unroll
        for (uint16_t c = tc; c < gpu_cols; c += blockDim.x * gridDim.x)
            v[AT(gpu_cols, tr, c)] = calc_mandelbrot(tr, c);
    }

}

int main() {

    hipSetDevice(0);

    rgb_t *gpu_vct;
    hipMalloc(&gpu_vct, sizeof(rgb_t) * cols * rows);
    hipMalloc(&gpu_vct, aligned_bsize_calc<sizeof(rgb_t)>(sizeof(rgb_t) * cols * rows));

    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);

    dim3 threadsPerBlock((int)sqrt(maxThreadsPerBlock), (int)sqrt(maxThreadsPerBlock));
    dim3 blocksDim(
            (cols + threadsPerBlock.x - 1) / threadsPerBlock.x, // force the ""alignment"" to a multiple of threadsPerBlock
            (rows + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    kernel<<<blocksDim, threadsPerBlock>>>(gpu_vct, cols * rows);
    hipDeviceSynchronize(); // wait for gpu

    PPM img{cols, rows};
    hipMemcpy(img.unwrap(), gpu_vct, sizeof(rgb_t) * img.width() * img.height(), hipMemcpyDeviceToHost);
    hipMemcpy(img.unwrap(), gpu_vct, aligned_bsize_calc<sizeof(rgb_t)>(sizeof(rgb_t) * cols * rows), hipMemcpyDeviceToHost);

    //memset(img.unwrap(), 0xff, aligned_bsize_calc<sizeof(rgb_t)>(sizeof(rgb_t) * cols * rows));
    img.write_file_content<PPM::Format::PPM3>("test.ppm3");
    img.write_file_content<PPM::Format::PPM6>("test.ppm6");

    PPM x{3, 2};
    //memset(x.unwrap(), 0xff, 3*2);

    x(0,0) = {255, 0,   0};
    x(0,1) = {0,   255, 0};
    x(0,2) = {0,   0,   255};

    x(1,0) = {255, 255, 0};
    x(1,1) = {255, 255, 255};
    x(1,2) = {0,   0,   0};

    x.write_file_content<PPM::Format::PPM6>("color.ppm6");
    x.write_file_content<PPM::Format::PPM3>("color.ppm3");

    hipFree(gpu_vct);
    hipDeviceReset();

    return 0;
}