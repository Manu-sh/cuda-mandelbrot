#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <ppm3.hpp>
#include <common.hpp>

#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <memalign/al_allocator.hpp>

inline constexpr uint16_t cols = 1920, rows = 1080;
//inline constexpr uint16_t cols = 4096, rows = 3112;
static_assert(
        ((rows-1) * cols) <= 0xff'ff'ff,
        "ATfast(cols, r_idx, c_idx) -> __umul24(r_idx, cols) does not support more than 24 bit"
);


// https://stackoverflow.com/questions/16119923/using-constants-with-cuda
__device__ inline constexpr uint16_t ixsize = rows, gpu_rows = rows, iysize = cols, gpu_cols = cols, max_i = 1000;
__device__ inline constexpr float cxmin = -2.5f, cxmax = 2.5f, cymin = -2.5f, cymax = 2.5f;

__device__ rgb_t calc_mandelbrot(uint16_t ix, uint16_t iy) {

    using thrust::complex, thrust::abs, thrust::cos;

    complex<float> c{
            cxmin + ix / (ixsize-1.0f) * (cxmax - cxmin),
            cymin + iy / (iysize-1.0f) * (cymax - cymin)
    }, z = 0;

    c *= c * c; // c = pow(c, 3);

    uint16_t i;
    for (i = 0; i < max_i && abs(z) < 4.0f; ++i)
        z = cos(z * c); // z = cos(z * pow(c, 3));

    // GPU handle float truncation differently from CPU, making round-toward-zero instead of round-to-nearest.
    // that took me a lot debugging to figure why the image generated was so different than another generated
    // by the same code but running on cpu instead of gpu

    //const uint8_t col = (uint8_t)(int)(z.real() * 1);                 // -> works but is signed integer overflow which is UB
    //const uint8_t col = min(255, max(0, (unsigned)(z.real() * 255))); // -> different result
    //const uint8_t col = (uint8_t)lround(z.real() * 1);                // ok
    const uint8_t col = (uint8_t)__float2int_rn(z.real() * 1);          // same of calling std::lround(float) but cuda specific
    return (i == max_i) ? rgb_t{0,0,0} : rgb_t{col,col,col};
}


__global__ void kernel(rgb_t *const v, uint32_t len) {

          uint16_t tr = blockIdx.y * blockDim.y + threadIdx.y;
    const uint16_t tc = blockIdx.x * blockDim.x + threadIdx.x;

    #pragma unroll
    for (; tr < gpu_rows; tr += blockDim.y * gridDim.y) {
        #pragma unroll
        for (uint16_t c = tc; c < gpu_cols; c += blockDim.x * gridDim.x)
            v[AT(gpu_cols, tr, c)] = calc_mandelbrot(tr, c);
    }

}

#if 0
int main() {

    hipSetDevice(0);

    rgb_t *gpu_vct;
    hipMalloc(&gpu_vct, sizeof(rgb_t) * cols * rows);

    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);

    dim3 threadsPerBlock((int)sqrt(maxThreadsPerBlock), (int)sqrt(maxThreadsPerBlock));
    dim3 blocksDim(
            (cols + threadsPerBlock.x - 1) / threadsPerBlock.x, // force the ""alignment"" to a multiple of threadsPerBlock
            (rows + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    kernel<<<blocksDim, threadsPerBlock>>>(gpu_vct, cols * rows);
    hipDeviceSynchronize(); // wait for gpu

    PPM3 img{cols, rows};
    hipMemcpy(img.unwrap(), gpu_vct, sizeof(PPM3::pixel_type) * img.width() * img.height(), hipMemcpyDeviceToHost);
    img.write_file_content("test.ppm");

    hipFree(gpu_vct);
    hipDeviceReset();

    return 0;
}
#else
#include <iostream>
#include <unistd.h>

using std::cout, std::endl;
int main() {

    struct __attribute__((__packed__)) big_t {
        big_t() {
            p = malloc(113);
            cout << "constructor called, malloc pointer " << p << '\n';
        }
        ~big_t() {
            free(p);
            cout << "destructor called, free pointer " << p << '\n';
        }
        void *p = nullptr;
        char buf[11];
    };

    /*
    cout << sizeof(big_t) << endl;
    Matrix1D<big_t, 4> mtx{3, 2};
    //Matrix1D<char, 3> mtx2{3, 3}; // should raise an error at compilation time
    Matrix1D<char, 1> mtx2{3, 3};
     */

    ///std::basic_string<char, std::char_traits<char>, ::al_allocator<char, 32> > s;
    //std::basic_string<char, std::char_traits<char>, std::allocator<char> > s;
    //std::basic_string<char, std::char_traits<char>, ::fuck_allocator<char> > s;

    std::basic_string<char, std::char_traits<char>, ::fuck_allocator<char> > s;
    for (int i = 0, x = 1920 * 1080 * 2; i < x; ++i) {
        s += std::string("ciao mondo");
    }

    return 0;
}
#endif