#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#include <pnm/ppm/PPM.hpp>
#include <pnm/pgm/PGM.hpp>
#include <pnm/ppm/PPM.hpp>

#include <pnm/common.hpp>
#include <pnm/memalign/utils.hpp>

// https://stackoverflow.com/questions/2151084/map-a-2d-array-onto-a-1d-array
constexpr FORCED(inline) uint32_t AT(uint16_t cols, uint16_t r, uint16_t c) { // cols = width = x_size
    return r * cols + c;
}

using namespace pnm;

inline constexpr uint16_t cols = 1920, rows = 1080;
//inline constexpr uint16_t cols = 4096, rows = 3112;
static_assert(
        ((rows-1) * cols) <= 0xff'ff'ff,
        "ATfast(cols, r_idx, c_idx) -> __umul24(r_idx, cols) does not support more than 24 bit"
);


// https://stackoverflow.com/questions/16119923/using-constants-with-cuda
__device__ inline constexpr uint16_t ixsize = rows, gpu_rows = rows, iysize = cols, gpu_cols = cols, max_i = 1000;
__device__ inline constexpr float cxmin = -2.5f, cxmax = 2.5f, cymin = -2.5f, cymax = 2.5f;


template <typename Pixel>
__device__ Pixel calc_mandelbrot(uint16_t ix, uint16_t iy) {

    using thrust::complex, thrust::abs, thrust::cos;

    complex<float> c{
            cxmin + ix / (ixsize-1.0f) * (cxmax - cxmin),
            cymin + iy / (iysize-1.0f) * (cymax - cymin)
    }, z = 0;

    c *= c * c; // c = pow(c, 3);

    uint16_t i;
    for (i = 0; i < max_i && abs(z) < 4.0f; ++i)
        z = cos(z * c); // z = cos(z * pow(c, 3));

    // GPU handle float truncation differently from CPU, making round-toward-zero instead of round-to-nearest.
    // that took me a lot debugging to figure why the image generated was so different than another generated
    // by the same code but running on cpu instead of gpu

    //const uint8_t col = (uint8_t)(int)(z.real() * 1);                 // -> works but is signed integer overflow which is UB
    //const uint8_t col = min(255, max(0, (unsigned)(z.real() * 255))); // -> different result
    //const uint8_t col = (uint8_t)lround(z.real() * 1);                // ok
    const uint8_t col = (uint8_t)__float2int_rn(z.real() * 1);          // same of calling std::lround(float) but cuda specific
    return (i == max_i) ? Pixel{0,0,0} : Pixel{col,col,col};
}


template<typename Pixel>
__global__ void kernel(Pixel *const v, uint32_t len) {

    uint16_t tr = blockIdx.y * blockDim.y + threadIdx.y;
    const uint16_t tc = blockIdx.x * blockDim.x + threadIdx.x;

#pragma unroll
    for (; tr < gpu_rows; tr += blockDim.y * gridDim.y) {
#pragma unroll
        for (uint16_t c = tc; c < gpu_cols; c += blockDim.x * gridDim.x)
            v[AT(gpu_cols, tr, c)] = calc_mandelbrot<Pixel>(tr, c);
    }

}

int main() {
#if 1
    PPM<pnm::rgb<pnm::BIT_8>> x{3, 2};

    x(0,0) = {255, 0,   0};
    x(0,1) = {0,   255, 0};
    x(0,2) = {0,   0,   255};

    x(1,0) = {255, 255, 0};
    x(1,1) = {255, 255, 255};
    x(1,2) = {0,   0,   0};

    x.write_file_content<pnm::Format::PPM3>("color.ppm3");
    x.write_file_content<pnm::Format::PPM6>("color.ppm6");
    return 0;

    PGM<pnm::grayscale<pnm::BIT_8>> pgm{3, 2};

    pgm(0,0) = {255, 0,   0};
    pgm(0,1) = {0,   255, 0};
    pgm(0,2) = {0,   0,   255};

    pgm(1,0) = {255, 255, 0};
    pgm(1,1) = {255, 255, 255};
    pgm(1,2) = {0,   0,   0};

    pgm.write_file_content<pnm::Format::PGM2>("test.pgm2");
    pgm.write_file_content<pnm::Format::PGM5>("test.pgm5");
    return 0;
#endif

    //using pixel_t = pnm::grayscale<pnm::BIT_8>;
    using pixel_t = pnm::rgb<pnm::BIT_8>;
    hipSetDevice(0);

    pixel_t *gpu_vct;
    hipMalloc(&gpu_vct, sizeof(pixel_t) * cols * rows);

    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);

    dim3 threadsPerBlock((int)sqrt(maxThreadsPerBlock), (int)sqrt(maxThreadsPerBlock));
    dim3 blocksDim(
            (cols + threadsPerBlock.x - 1) / threadsPerBlock.x, // force the ""alignment"" to a multiple of threadsPerBlock
            (rows + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    kernel<<<blocksDim, threadsPerBlock>>>(gpu_vct, cols * rows);
    hipDeviceSynchronize(); // wait for gpu

#if 0
    PGM<pixel_t> img{cols, rows};
    hipMemcpy(img.unwrap(), gpu_vct, sizeof(pixel_t) * img.width() * img.height(), hipMemcpyDeviceToHost);
    img.write_file_content<pnm::Format::PGM2>("test.ppm2");
    img.write_file_content<pnm::Format::PGM5>("test.ppm5");
#else

    PPM<pixel_t> img{cols, rows};
    hipMemcpy(img.unwrap(), gpu_vct, sizeof(pixel_t) * img.width() * img.height(), hipMemcpyDeviceToHost);
    img.write_file_content<pnm::Format::PPM6>("test.ppm6");
    img.write_file_content<pnm::Format::PPM3>("test.ppm3");
#endif

    hipFree(gpu_vct);
    hipDeviceReset();
    return 0;
}